#include "hip/hip_runtime.h"
#include "cuda_naive_convolve.h"


// Cuda scheduling parameters
#define BLOCK_SIDE 16
#define THREADS_PER_BLOCK (BLOCK_SIDE*BLOCK_SIDE)

// Helper macros
#define UPDIV(x,y) ((x+y-1)/y)

struct msg {
    float* padded_image;
    float* result_image;
    int padded_image_rows;
    int padded_image_cols;
    float* kernel;
};



msg cuda_device_setup(cv::Mat padded_image, cv::Mat kernel) {
    
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    // Fetch cuda device specifications and display
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, 0);
    std::string name = deviceProps.name;
    printf("Device : %s\n", deviceProps.name);
    printf("   SMMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    printf("---------------------------------------------------------\n");

    // Allocating memory to store the input image, kernel and output result
    // on the cuda device
    float* device_padded_image;
    float* device_kernel;
    float* device_result;
    int kernel_radius = kernel.rows/2;
    int kernel_size = kernel.rows*kernel.cols*sizeof(float);
    int result_rows = (padded_image.rows - 2*kernel_radius);
    int result_cols = (padded_image.cols - 2*kernel_radius);
    int result_size = result_rows * result_cols * sizeof(float);
    int padded_image_size = padded_image.rows * padded_image.cols * sizeof(float);
    hipMalloc( (void **)&device_padded_image, padded_image_size );
    hipMalloc( (void **)&device_kernel, kernel_size);
    hipMalloc( (void **)&device_result, result_size);

    cv::Mat result = cv::Mat::zeros(result_rows, result_cols, CV_32F);

    // Transferring image and kernel data to cuda device global memory
    hipMemcpy( device_padded_image, padded_image.data,
                                padded_image_size, hipMemcpyHostToDevice );
    hipMemcpy( device_kernel, kernel.data, kernel_size, hipMemcpyHostToDevice );
    hipMemcpy( device_result, result.data, result_size, hipMemcpyHostToDevice );

    msg device_ptrs;
    device_ptrs.padded_image = device_padded_image;
    device_ptrs.kernel = device_kernel;
    device_ptrs.padded_image_rows = padded_image.rows;
    device_ptrs.padded_image_cols = padded_image.cols;
    device_ptrs.result_image = device_result;

    return device_ptrs;
    
}

__global__ void kernel_convolve( int image_rows, int image_cols, int kernel_rows, int kernel_cols,
                                 int padded_img_rows, int padded_img_cols, int kernel_radius, 
                                 float kernel_sum,
                                 float* padded_image, float* kernel_arr, float* device_result )
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    //printf("(%d, %d)\n", row, col);   
    
    if(row<image_rows && col<image_cols)
    {
       int kernel_start_row = row - kernel_radius;
       int kernel_start_col = col - kernel_radius;
       
       for(int k_row=0; k_row<(kernel_rows); k_row++)
       {
           for(int k_col=0; k_col<(kernel_cols); k_col++)
           {
               //printf("result(%d,%d) += ",row,col);
               //printf("kernel(%d,%d) * ",k_row,k_col);
               //printf("padded_image(%d,%d)\n",kernel_radius+kernel_start_row+k_row,kernel_radius+kernel_start_col+k_col);
               device_result[(row*image_cols+col)] += 
               kernel_arr[((k_row)*kernel_cols+k_col)] *
               padded_image[((kernel_radius+kernel_start_row+k_row)*padded_img_cols + 
                                    kernel_radius+kernel_start_col+k_col)] / kernel_sum;
           }
       } 
    }
}


cv::Mat cuda_convolve(cv::Mat cpu_image, cv::Mat cpu_kernel) 
{   // TODO : pass by ref
    
    // Establish cudaThread grid
    dim3 block_dim(BLOCK_SIDE, BLOCK_SIDE);
    dim3 block_grid_dim(UPDIV(cpu_image.cols, block_dim.x),
                        UPDIV(cpu_image.rows, block_dim.y));
    //dim3 block_dim(1,2);
    //dim3 block_grid_dim(1,1);
    

    int kernel_radius = cpu_kernel.rows/2;
    
    // Pad image
    cv::Mat padded_image;
    cv::copyMakeBorder( cpu_image, padded_image, kernel_radius, kernel_radius, 
                        kernel_radius, kernel_radius, cv::BORDER_REPLICATE);

    //std::cout<<"padded = \n"<<padded_image;
    // Prepare the cuda device for kernel launch
    msg device_ptrs = cuda_device_setup(padded_image, cpu_kernel);
    float kernel_sum;
    for(int i=0; i<cpu_kernel.rows; i++)
    {
        for(int j=0; j<cpu_kernel.cols; j++)
        {
            kernel_sum += cpu_kernel.at<float>(i,j);
        }
    }

    // Start timer
    std::clock_t start;
    double duration;
    start = std::clock();

    // Launch Cuda convolve kernel
    kernel_convolve<<<block_grid_dim, block_dim>>>(cpu_image.rows, cpu_image.cols,
                                                   cpu_kernel.rows, cpu_kernel.cols,
                                                   device_ptrs.padded_image_rows,
                                                   device_ptrs.padded_image_cols,
                                                   kernel_radius, kernel_sum,
                                                   device_ptrs.padded_image,
                                                   device_ptrs.kernel,
                                                   device_ptrs.result_image
                                                   );
    
    // Wait for computation to complete across all cuda threads
    hipDeviceSynchronize();

    // Stopping timer
    duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
    std::cout<<"Time taken: "<< duration <<" seconds\n";

    // Copy result array back from CUDA memory
    int result_size = cpu_image.rows * cpu_image.cols * sizeof(float);
    float* result_arr = (float*)malloc(result_size);
    hipMemcpy(result_arr, device_ptrs.result_image, result_size, hipMemcpyDeviceToHost);
    
    // Convert float array to cv::Mat
    cv::Mat result = cv::Mat(cpu_image.rows, cpu_image.cols, CV_32F, result_arr);
    cv::Mat op_image;

    // Cast to CV_8U for display
    result.convertTo(op_image,CV_8U);
    
    return op_image;
}

int main(int argc, char** argv )
{
    if( argc != 2)
    {
        printf("usage: baseline_convolve <image_path> <kernel_path>\n");
        return -1;
    }

    // Reading image file
    cv::Mat gray_image = cv::imread(argv[1], 0 );
    cv::Mat image;
    gray_image.convertTo(image, CV_32F);
    //cv::Mat image = (cv::Mat_<float>(5,4) << 1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20);
    
    // Reading kernel file
    cv::Mat kernel = cv::Mat::ones(31, 31, CV_32F);;

    // Convolve
    cv::Mat result = cuda_convolve(image, kernel);
    //std::cout<<"Result = \n"<<result;
    
    // Displaying results
    cv::namedWindow( "Source Image", cv::WINDOW_AUTOSIZE );
    cv::namedWindow( "Convolved Image", cv::WINDOW_AUTOSIZE );
    cv::imshow("Source Image", gray_image);
    cv::imshow("Convolved Image", result);

    cv::waitKey(0);

    return 0;

}





