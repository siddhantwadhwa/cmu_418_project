#include "hip/hip_runtime.h"
#include "cuda_get_visual_words.h"

// Cuda scheduling parameters
#define BLOCK_SIDE 16
#define THREADS_PER_BLOCK (BLOCK_SIDE*BLOCK_SIDE)

// Helper macros
#define UPDIV(x,y) ((x+y-1)/y)



__global__ 
void cuda_wordmap_kernel(int img_rows, int img_cols, int row_wise_fr_cols, int dict_rows, int dict_cols, float* row_wise_fr, float* dict, int* wordmap)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row>=img_rows || col>=img_cols)
        return;

    // Intializing min_idx and min_dist
    int min_idx = 0;
    float min_dist = 0;
    for(int j=0; j<row_wise_fr_cols; j++)
    {
        //float tmp = row_wise_fr[row*row_wise_fr_cols+col][j] - dict[0][j];
        float tmp = row_wise_fr[(row*row_wise_fr_cols+col)*row_wise_fr_cols + j] - dict[0 + j];
        min_dist = sqrt(tmp*tmp);
    }

    for(int i=1; i<dict_rows; i++)
    {
        for(int j=0; j<row_wise_fr_cols; j++)
        {
            //float tmp = row_wise_fr[row*row_wise_fr_cols+col][j] - dict[i][j];
            float tmp = row_wise_fr[(row*row_wise_fr_cols+col)*row_wise_fr_cols + j] - dict[i*dict_cols + j];
            float dist = sqrt(tmp*tmp);
            if(dist<min_dist)
            {
                min_dist = dist;
                min_idx = i;
            }
        }
    }

    wordmap[row*img_cols + col] = min_idx;
    return;    
}

cv::Mat cuda_get_visual_words(cv::Mat img, cv::Mat dict, filter_bank fb)
{
    // Get filter responses
    std::vector<cv::Mat> filter_responses = extractFilterResponses(img, fb,0);
    
    // Reshape filter_responses, so that each pixel gets a row of 3n dimensions
    cv::Mat row_wise_filter_response = cv::Mat::zeros(img.rows*img.cols, filter_responses.size(), CV_32F);
    for(int row=0; row<img.rows; row++)
    {
        for(int col=0; col<img.cols; col++)
        {
            for(int i=0; i<filter_responses.size(); i++)
            {
                row_wise_filter_response.at<float>(row*img.cols+col,i) = 
                filter_responses[i].at<float>(row,col);
        
            }
        }
    }
    
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    // Fetch cuda device specifications and display
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, 0);
    std::string name = deviceProps.name;
    printf("Device : %s\n", deviceProps.name);
    printf("   SMMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    printf("---------------------------------------------------------\n");

    // Declare pointers to device data
    float* device_row_wise_filter_responses;
    float* device_dict;
    int* device_wordmap;

    // Compute sizes of data to be transferred to device
    int row_wise_filter_responses_size = row_wise_filter_response.rows * row_wise_filter_response.cols * sizeof(float);
    int dict_size = dict.rows*dict.cols*sizeof(float);
    int wordmap_size = img.rows*img.cols*sizeof(int);
    
    // Allocate device memory
    hipMalloc( (void **)&device_row_wise_filter_responses, row_wise_filter_responses_size );
    hipMalloc( (void **)&device_dict, dict_size );
    hipMalloc( (void **)&device_wordmap, wordmap_size );

    // Copy over data to device
    hipMemcpy( device_row_wise_filter_responses, row_wise_filter_response.data, row_wise_filter_responses_size, hipMemcpyHostToDevice );
    hipMemcpy( device_dict, dict.data, dict_size, hipMemcpyHostToDevice );

    // Establish cudaThread grid
    dim3 block_dim(BLOCK_SIDE, BLOCK_SIDE);
    dim3 block_grid_dim(UPDIV(img.cols, block_dim.x),
                        UPDIV(img.rows, block_dim.y));

    // Launch cuda kernel
    cuda_wordmap_kernel<<<block_grid_dim, block_dim>>>(img.rows, img.cols, row_wise_filter_response.cols, dict.rows, dict.cols,
                                                       device_row_wise_filter_responses, device_dict, device_wordmap);
    hipDeviceSynchronize();

    int* wordmap_arr = (int*)malloc(wordmap_size);
    hipMemcpy(wordmap_arr, device_wordmap, dict_size, hipMemcpyDeviceToHost);
    cv::Mat wordmap = cv::Mat(img.rows, img.cols, CV_32S, wordmap_arr);

    return wordmap;
}

